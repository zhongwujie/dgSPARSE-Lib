// file: spmm.cu.cc
//
// Using cusparse API to test SpMM performance.
//  author: guyue huang
//  date  : 2021/10/13
// compile: nvcc version >=11.0

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM (>= v11.0) or hipsparseScsrmm
#include <stdio.h>
#include <stdlib.h>

#include <cstdlib> // std::rand(), RAND_MAX
#include <vector>
#include <string>
#include <map>

#include "../../src/ge-spmm/gespmm.h" // gespmmCsrSpMM()
#include "../util/sp_util.hpp"        // read_mtx

gespmmAlg_t GetAlgName(std::string name){
  gespmmAlg_t alg;
  switch (name) {
    case "SEQREDUCE_ROWBALANCE":
      alg = GESPMM_ALG_SEQREDUCE_ROWBALANCE;
      break;
    case "PARREDUCE_ROWBALANCE":
      alg = GESPMM_ALG_PARREDUCE_ROWBALANCE;
      break;
    case "SEQREDUCE_NNZBALANCE":
      alg = GESPMM_ALG_SEQREDUCE_NNZBALANCE;
      break;
    case "PARREDUCE_NNZBALANCE":
      alg = GESPMM_ALG_PARREDUCE_NNZBALANCE;
      break;
    case "SEQREDUCE_ROWBALANCE_NON_TRANSPOSE":
      alg = GESPMM_ALG_SEQREDUCE_ROWBALANCE_NON_TRANSPOSE;
      break;
    case "PARREDUCE_ROWBALANCE_NON_TRANSPOSE":
      alg = GESPMM_ALG_PARREDUCE_ROWBALANCE_NON_TRANSPOSE;
      break;
    case "SEQREDUCE_NNZBALANCE_NON_TRANSPOSE":
      alg = GESPMM_ALG_SEQREDUCE_NNZBALANCE_NON_TRANSPOSE;
      break;
    case "PARREDUCE_NNZBALANCE_NON_TRANSPOSE":
      alg = GESPMM_ALG_PARREDUCE_NNZBALANCE_NON_TRANSPOSE;
      break;
    case "ROWCACHING_ROWBALANCE":
      alg = GESPMM_ALG_ROWCACHING_ROWBALANCE;
      break;
    case "ROWCACHING_NNZBALANCE":
      alg = GESPMM_ALG_ROWCACHING_NNZBALANCE;
      break;
    default:
      break;
  }
  return alg;
} 

int main(int argc, const char **argv) {
  /// check command-line argument

  if (argc < 2) {
    printf("Require command-line argument: name of the sparse matrix file in "
           ".mtx format.\n");
    return EXIT_FAILURE;
  }

  //
  // Load sparse matrix
  //

  int M;                               // number of A-rows
  int K;                               // number of A-columns
  int nnz;                             // number of non-zeros in A
  std::vector<int> csr_indptr_buffer;  // buffer for indptr array in CSR format
  std::vector<int> csr_indices_buffer; // buffer for indices (column-ids) array
                                       // in CSR format
  // load sparse matrix from mtx file
  read_mtx_file(argv[1], M, K, nnz, csr_indptr_buffer, csr_indices_buffer);
  printf("Finish reading matrix %d rows, %d columns, %d nnz. \nIgnore original "
         "values and use randomly generated values.\n",
         M, K, nnz);

  // Create GPU arrays
  int N = 128; // number of B-columns
  if (argc > 2) {
    N = atoi(argv[2]);
  }
  assert(
      N > 0 &&
      "second command-line argument is number of B columns, should be >0.\n");
  
  gespmmAlg_t alg;
  bool cusparse_bit = false;
  std::string alg_name = "";
  if (argc > 3){
    alg_name = argv[3];
    if(alg_name == "CUSPARSE")  cusparse_bit = true;
    else alg = GetAlgName(alg_name);
  }
  else alg = GESPMM_ALG_SEQREDUCE_ROWBALANCE;

  float *B_h = NULL, *C_h = NULL, *csr_values_h = NULL, *C_ref = NULL;
  float *B_d = NULL, *C_d = NULL, *csr_values_d = NULL;
  int *csr_indptr_d = NULL, *csr_indices_d = NULL;

  B_h = (float *)malloc(sizeof(float) * K * N);
  C_h = (float *)malloc(sizeof(float) * M * N);
  C_ref = (float *)malloc(sizeof(float) * M * N);
  csr_values_h = (float *)malloc(sizeof(float) * nnz);
  if (!B_h || !C_h || !C_ref || !csr_values_h) {
    printf("Host allocation failed.\n");
    return EXIT_FAILURE;
  }

  fill_random(csr_values_h, nnz);
  fill_random(B_h, K * N);

  CUDA_CHECK(hipMalloc((void **)&B_d, sizeof(float) * K * N));
  CUDA_CHECK(hipMalloc((void **)&C_d, sizeof(float) * M * N));
  CUDA_CHECK(hipMalloc((void **)&csr_values_d, sizeof(float) * nnz));
  CUDA_CHECK(hipMalloc((void **)&csr_indptr_d, sizeof(int) * (M + 1)));
  CUDA_CHECK(hipMalloc((void **)&csr_indices_d, sizeof(int) * nnz));

  CUDA_CHECK(
      hipMemcpy(B_d, B_h, sizeof(float) * K * N, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(C_d, 0x0, sizeof(float) * M * N));
  CUDA_CHECK(hipMemcpy(csr_values_d, csr_values_h, sizeof(float) * nnz,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(csr_indptr_d, csr_indptr_buffer.data(),
                        sizeof(int) * (M + 1), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(csr_indices_d, csr_indices_buffer.data(),
                        sizeof(int) * nnz, hipMemcpyHostToDevice));

  //
  // Run Cusparse-SpMM and check result
  //
  if(cusparse_bit){
    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t csrDescr;
    hipsparseDnMatDescr_t dnMatInputDescr, dnMatOutputDescr;
    float alpha = 1.0f, beta = 0.0f;

    CUSPARSE_CHECK(hipsparseCreate(&handle));

    // creating sparse csr matrix
    CUSPARSE_CHECK(hipsparseCreateCsr(
        &csrDescr, M, K, nnz, csr_indptr_d, csr_indices_d, csr_values_d,
        HIPSPARSE_INDEX_32I, // index 32-integer for indptr
        HIPSPARSE_INDEX_32I, // index 32-integer for indices
        HIPSPARSE_INDEX_BASE_ZERO,
        HIP_R_32F // datatype: 32-bit float real number
        ));

    // creating dense matrices
    CUSPARSE_CHECK(hipsparseCreateDnMat(&dnMatInputDescr, K, N, N, B_d, HIP_R_32F,
                                      HIPSPARSE_ORDER_ROW));
    CUSPARSE_CHECK(hipsparseCreateDnMat(&dnMatOutputDescr, M, N, N, C_d,
                                      HIP_R_32F, HIPSPARSE_ORDER_ROW));

    // allocate workspace buffer
    size_t workspace_size;
    CUSPARSE_CHECK(hipsparseSpMM_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, csrDescr, dnMatInputDescr,
        &beta, dnMatOutputDescr, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT,
        &workspace_size));

    void *workspace = NULL;
    CUDA_CHECK(hipMalloc(&workspace, workspace_size));

    // run SpMM
    CUSPARSE_CHECK(hipsparseSpMM(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE, // opA
                                HIPSPARSE_OPERATION_NON_TRANSPOSE, // opB
                                &alpha, csrDescr, dnMatInputDescr, &beta,
                                dnMatOutputDescr, HIP_R_32F,
                                HIPSPARSE_SPMM_ALG_DEFAULT, workspace));

    CUDA_CHECK(
        hipMemcpy(C_h, C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost));

    spmm_reference_host<int, float>(M, N, K, csr_indptr_buffer.data(),
                                    csr_indices_buffer.data(), csr_values_h, B_h,
                                    C_ref);

    bool correct = check_result<float>(M, N, C_h, C_ref);

    //
    // Benchmark Cusparse-SpMM performance
    //

    if (correct) {
      GpuTimer gpu_timer;
      int warmup_iter = 10;
      int repeat_iter = 100;
      for (int iter = 0; iter < warmup_iter + repeat_iter; iter++) {
        if (iter == warmup_iter) {
          gpu_timer.start();
        }

        hipsparseSpMM(handle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE, // opA
                    HIPSPARSE_OPERATION_NON_TRANSPOSE, // opB
                    &alpha, csrDescr, dnMatInputDescr, &beta, dnMatOutputDescr,
                    HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, workspace);
      }
      gpu_timer.stop();

      float kernel_dur_msecs = gpu_timer.elapsed_msecs() / repeat_iter;

      float MFlop_count = (float)nnz / 1e6 * N * 2;

      float gflops = MFlop_count / kernel_dur_msecs;

      printf("[Cusparse] Report: spmm A(%d x %d) * B(%d x %d) sparsity %f "
            "(nnz=%d) \n Time %f (ms), Throughput %f (gflops).\n",
            M, K, K, N, (float)nnz / M / K, nnz, kernel_dur_msecs, gflops);
    }
  }
  else{
    
    SpMatCsrDescr_t spmatA{M, K, nnz, csr_indptr_d, csr_indices_d, csr_values_d};

    //
    // Run GE-SpMM and check result
    //

    CUDA_CHECK(hipMemset(C_d, 0x0, sizeof(float) * M * N));

    gespmmCsrSpMM(spmatA, B_d, N, C_d, true, alg);

    hipDeviceSynchronize();
    CUDA_CHECK(
        hipMemcpy(C_h, C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost));

    spmm_reference_host<int, float>(M, N, K, csr_indptr_buffer.data(),
                                    csr_indices_buffer.data(), csr_values_h,
                                    B_h, C_ref);

    bool correct = check_result<float>(M, N, C_h, C_ref);

    if (correct) {
      // benchmark GE-SpMM performance

      GpuTimer gpu_timer;
      int warmup_iter = 10;
      int repeat_iter = 100;
      for (int iter = 0; iter < warmup_iter + repeat_iter; iter++) {
        if (iter == warmup_iter) {
          gpu_timer.start();
        }

        gespmmCsrSpMM(spmatA, B_d, N, C_d, true, alg);
      }
      gpu_timer.stop();

      float kernel_dur_msecs = gpu_timer.elapsed_msecs() / repeat_iter;

      float MFlop_count = (float)nnz / 1e6 * N * 2;

      float gflops = MFlop_count / kernel_dur_msecs;

      printf("[GE-SpMM][Alg: %s] Report: spmm A(%d x %d) * B(%d x %d) sparsity "
              "%f (nnz=%d) \n Time %f (ms), Throughput %f (gflops).\n",
              alg_name, M, K, K, N, (float)nnz / M / K, nnz, kernel_dur_msecs,
              gflops);
    }
  }

  /// free memory

  if (B_h)
    free(B_h);
  if (C_h)
    free(C_h);
  if (C_ref)
    free(C_ref);
  if (csr_values_h)
    free(csr_values_h);
  if (B_d)
    CUDA_CHECK(hipFree(B_d));
  if (C_d)
    CUDA_CHECK(hipFree(C_d));
  if (csr_values_d)
    CUDA_CHECK(hipFree(csr_values_d));
  if (csr_indptr_d)
    CUDA_CHECK(hipFree(csr_indptr_d));
  if (csr_indices_d)
    CUDA_CHECK(hipFree(csr_indices_d));
  if (workspace)
    CUDA_CHECK(hipFree(workspace));

  // destroy matrix/vector descriptors
  CUSPARSE_CHECK(hipsparseDestroyDnMat(dnMatInputDescr));
  CUSPARSE_CHECK(hipsparseDestroyDnMat(dnMatOutputDescr));
  CUSPARSE_CHECK(hipsparseDestroySpMat(csrDescr));
  CUSPARSE_CHECK(hipsparseDestroy(handle));
  return EXIT_SUCCESS;
}
